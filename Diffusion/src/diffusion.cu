#include "hip/hip_runtime.h"
/****************************************************************************\
*      --- Practical Course: GPU Programming in Computer Vision ---
*
* time:    winter term 2012/13 / March 11-18, 2013
*
* project: diffusion
* file:    diffusion.cu
*
* 
\******* PLEASE ENTER YOUR CORRECT STUDENT LOGIN, NAME AND ID BELOW *********/
const char* studentLogin = "p116";
const char* studentName  = "Arash Bakhtiari";
const int   studentID    = 03625141;
/****************************************************************************\
*
* In this file the following methods have to be edited or completed:
*
* diffuse_linear_isotrop_shared(const float  *d_input, ... )
* diffuse_linear_isotrop_shared(const float3 *d_input, ... )
* diffuse_nonlinear_isotrop_shared(const float  *d_input, ... )
* diffuse_nonlinear_isotrop_shared(const float3 *d_input, ... )
* compute_tv_diffusivity_shared
* compute_tv_diffusivity_joined_shared
* compute_tv_diffusivity_separate_shared
* jacobi_shared(float  *d_output, ... )
* jacobi_shared(float3 *d_output, ... )
* sor_shared(float  *d_output, ... )
* sor_shared(float3 *d_output, ... )
*
\****************************************************************************/


#define DIFF_BW 16
#define DIFF_BH 16

#define TV_EPSILON 0.1f


#include "diffusion.cuh"



const char* getStudentLogin() { return studentLogin; };
const char* getStudentName()  { return studentName; };
int         getStudentID()    { return studentID; };
bool checkStudentData() { return strcmp(studentLogin, "p010") != 0 && strcmp(studentName, "John Doe") != 0 && studentID != 1234567; };
bool checkStudentNameAndID() { return strcmp(studentName, "John Doe") != 0 && studentID != 1234567; };


//----------------------------------------------------------------------------
// Linear Diffusion
//----------------------------------------------------------------------------


// mode 0 gray: linear diffusion
__global__ void diffuse_linear_isotrop_shared(
  const float *d_input,
  float *d_output,
  float timeStep, 
  int nx, int ny,
  size_t pitch)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;
  const int idx = y*pitch + x;
  //d_output[idx] = 0;
  __shared__ float u[DIFF_BW+2][DIFF_BH+2];

  // load data into shared memory
  if (x < nx && y < ny) {

    u[tx][ty] = d_input[idx];

    if (x == 0)  u[0][ty] = u[tx][ty];
    else if (threadIdx.x == 0) u[0][ty] = d_input[idx-1];
    if (x == nx-1) u[tx+1][ty] = u[tx][ty];
    else if (threadIdx.x == blockDim.x-1) u[tx+1][ty] = d_input[idx+1];

    if (y == 0)  u[tx][0] = u[tx][ty];
    else if (threadIdx.y == 0) u[tx][0] = d_input[idx-pitch];
    if (y == ny-1) u[tx][ty+1] = u[tx][ty];
    else if (threadIdx.y == blockDim.y-1) u[tx][ty+1] = d_input[idx+pitch];
  }

  __syncthreads();

  // ### implement me ###
  if (x < nx && y < ny) {
	d_output[idx] = u[tx][ty] + timeStep * ( u[tx + 1][ty]
			+ u[tx - 1][ty] + u[tx][ty + 1]
			+ u[tx][ty - 1] - 4 * u[tx][ty]);
  }
}



// mode 0 interleaved: linear diffusion
__global__ void diffuse_linear_isotrop_shared
(
 const float3 *d_input,
 float3 *d_output,
 float timeStep,
 int nx, int ny,
 size_t pitchBytes
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;
  const char* imgP = (char*)d_input + y*pitchBytes + x*sizeof(float3);

  __shared__ float3 u[DIFF_BW+2][DIFF_BH+2];
  float3 imgValue;

  // load data into shared memory
  if (x < nx && y < ny) {

    imgValue = *( (float3*)imgP );
    u[tx][ty] = imgValue;

    if (x == 0)  u[0][ty] = imgValue;
    else if (threadIdx.x == 0) u[0][ty] = *( ((float3*)imgP)-1 );
    if (x == nx-1) u[tx+1][ty] = imgValue;
    else if (threadIdx.x == blockDim.x-1) u[tx+1][ty] = *( ((float3*)imgP)+1 );

    if (y == 0)  u[tx][0] = imgValue;
    else if (threadIdx.y == 0) u[tx][0] = *( (float3*)(imgP-pitchBytes) );
    if (y == ny-1) u[tx][ty+1] = imgValue;
    else if (threadIdx.y == blockDim.y-1) u[tx][ty+1] = *( (float3*)(imgP+pitchBytes) );
  }

  __syncthreads();

  float3 tmpValue;
  tmpValue.x = u[tx][ty].x + timeStep * (u[tx + 1][ty].x + u[tx - 1][ty].x
			+ u[tx][ty + 1].x + u[tx][ty - 1].x - 4 * u[tx][ty].x);
  
  tmpValue.y = u[tx][ty].y + timeStep * (u[tx + 1][ty].y + u[tx - 1][ty].y
			+ u[tx][ty + 1].y + u[tx][ty - 1].y - 4 * u[tx][ty].y);

  tmpValue.z = u[tx][ty].z + timeStep * (u[tx + 1][ty].z + u[tx - 1][ty].z
			+ u[tx][ty + 1].z + u[tx][ty - 1].z - 4 * u[tx][ty].z);
  if (x < nx && y < ny) 
  *((float3*)(((char*)d_output) + y*pitchBytes) + x) = tmpValue;
  
}




//----------------------------------------------------------------------------
// Non-linear Diffusion - explicit scheme
//----------------------------------------------------------------------------



// mode 1 gray: nonlinear diffusion
__global__ void diffuse_nonlinear_isotrop_shared
(
 const float *d_input,
 const float *d_diffusivity,
 float *d_output,
 float timeStep,
 int   nx,
 int   ny,
 size_t   pitch
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;
  const int idx = y*pitch + x;

  __shared__ float u[DIFF_BW+2][DIFF_BH+2];
  __shared__ float g[DIFF_BW+2][DIFF_BH+2];


  // load data into shared memory
  if (x < nx && y < ny) {
    u[tx][ty] = d_input[idx];
    g[tx][ty] = d_diffusivity[idx];

    if (x == 0) {
      u[0][ty] = u[tx][ty];
      g[0][ty] = g[tx][ty];
    }
    else if (threadIdx.x == 0) {
      u[0][ty] = d_input[idx-1];
      g[0][ty] = d_diffusivity[idx-1];
    }
      
    if (x == nx-1) {
      u[tx+1][ty] = u[tx][ty];
      g[tx+1][ty] = g[tx][ty];
    }
    else if (threadIdx.x == blockDim.x-1) {
      u[tx+1][ty] = d_input[idx+1];
      g[tx+1][ty] = d_diffusivity[idx+1];
    }


    if (y == 0) {
      u[tx][0] = u[tx][ty];
      g[tx][0] = g[tx][ty];
    }
    else if (threadIdx.y == 0) {
      u[tx][0] = d_input[idx-pitch];
      g[tx][0] = d_diffusivity[idx-pitch];
    }
      
    if (y == ny-1) {
      u[tx][ty+1] = u[tx][ty];
      g[tx][ty+1] = g[tx][ty];
    } 
    else if (threadIdx.y == blockDim.y-1) {
      u[tx][ty+1] = d_input[idx+pitch];
      g[tx][ty+1] = d_diffusivity[idx+pitch];
    }
  }

  __syncthreads();

  float phiR = 0.5 * (g[tx+1][ty] + g[tx][ty]);
  float phiL = 0.5 * (g[tx-1][ty] + g[tx][ty]);
  float phiU = 0.5 * (g[tx][ty+1] + g[tx][ty]);
  float phiD = 0.5 * (g[tx][ty-1] + g[tx][ty]);
  
  // ### implement me ###
	if (x < nx && y < ny) {
		d_output[idx] = u[tx][ty] + timeStep * (
						u[tx + 1][ty]*phiR +
						u[tx - 1][ty]*phiL + 
						u[tx][ty + 1]*phiU + 
						u[tx][ty - 1]*phiD - 
						u[tx][ty]*(phiR+phiL+phiU+phiD) );
	}

}



// mode 1 interleaved: nonlinear diffusion
__global__ void diffuse_nonlinear_isotrop_shared
(
 const float3 *d_input,
 const float3 *d_diffusivity,
 float3 *d_output,
 float timeStep,
 int   nx,
 int   ny,
 size_t   pitchBytes
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;
  const char* imgP = (char*)d_input + y*pitchBytes + x*sizeof(float3);
  const char* diffP = (char*)d_diffusivity + y*pitchBytes + x*sizeof(float3);

  __shared__ float3 u[DIFF_BW+2][DIFF_BH+2];
  __shared__ float3 g[DIFF_BW+2][DIFF_BH+2];
  float3 value;


  // load data into shared memory
  if (x < nx && y < ny) {
    u[tx][ty] = *( (float3*)imgP );
    g[tx][ty] = *( (float3*)diffP );

    if (x == 0) {
      u[0][ty] = u[tx][ty];
      g[0][ty] = g[tx][ty];
    }
    else if (threadIdx.x == 0) {
      u[0][ty] = *( ((float3*)imgP)-1 );
      g[0][ty] = *( ((float3*)diffP)-1 );
    }
    if (x == nx-1) {
      u[tx+1][ty] = u[tx][ty];
      g[tx+1][ty] = g[tx][ty];
    } 
    else if (threadIdx.x == blockDim.x-1) {
      u[tx+1][ty] = *( ((float3*)imgP)+1 );
      g[tx+1][ty] = *( ((float3*)diffP)+1 );
    }

    if (y == 0) {
      u[tx][0] = u[tx][ty];
      g[tx][0] = g[tx][ty];
    } 
    else if (threadIdx.y == 0) {
      u[tx][0] = *( (float3*)(imgP-pitchBytes) );
      g[tx][0] = *( (float3*)(diffP-pitchBytes) );
    }
    if (y == ny-1) {
      u[tx][ty+1] = u[tx][ty];
      g[tx][ty+1] = g[tx][ty];
    }
    else if (threadIdx.y == blockDim.y-1) {
      u[tx][ty+1] = *( (float3*)(imgP+pitchBytes) );
      g[tx][ty+1] = *( (float3*)(diffP+pitchBytes) );
    }
  }

  __syncthreads();
  float3 phiR,phiL,phiU, phiD;
  
    phiR.x = 0.5 * (g[tx+1][ty].x+ g[tx][ty].x);
	phiL.x = 0.5 * (g[tx-1][ty].x+ g[tx][ty].x);
	phiU.x = 0.5 * (g[tx][ty+1].x+ g[tx][ty].x);
	phiD.x = 0.5 * (g[tx][ty-1].x+ g[tx][ty].x);
	
    phiR.y= 0.5 * (g[tx+1][ty].y+ g[tx][ty].y);
	phiL.y= 0.5 * (g[tx-1][ty].y+ g[tx][ty].y);
	phiU.y= 0.5 * (g[tx][ty+1].y+ g[tx][ty].y);
	phiD.y= 0.5 * (g[tx][ty-1].y+ g[tx][ty].y);
	
    phiR.z= 0.5 * (g[tx+1][ty].z+ g[tx][ty].z);
	phiL.z= 0.5 * (g[tx-1][ty].z+ g[tx][ty].z);
	phiU.z= 0.5 * (g[tx][ty+1].z+ g[tx][ty].z);
	phiD.z= 0.5 * (g[tx][ty-1].z+ g[tx][ty].z);
	
	// ### implement me ###
	float3 res;
	if (x < nx && y < ny) {
		res.x = u[tx][ty].x + timeStep * (
						u[tx + 1][ty].x*phiR.x +
						u[tx - 1][ty].x*phiL.x + 
						u[tx][ty + 1].x*phiU.x + 
						u[tx][ty - 1].x*phiD.x - 
						u[tx][ty].x*(phiR.x+phiL.x+phiU.x+phiD.x) );
		
		res.y = u[tx][ty].y + timeStep * (
						u[tx + 1][ty].y*phiR.y +
						u[tx - 1][ty].y*phiL.y + 
						u[tx][ty + 1].y*phiU.y + 
						u[tx][ty - 1].y*phiD.y - 
						u[tx][ty].y*(phiR.y+phiL.y+phiU.y+phiD.y) );
		
		res.z = u[tx][ty].z + timeStep * (
						u[tx + 1][ty].z*phiR.z +
						u[tx - 1][ty].z*phiL.z + 
						u[tx][ty + 1].z*phiU.z + 
						u[tx][ty - 1].z*phiD.z - 
						u[tx][ty].z*(phiR.z+phiL.z+phiU.z+phiD.z) );
		
		 *((float3*)(((char*)d_output) + y*pitchBytes) + x) = res;
	}
	
	
}


// diffusivity computation for modes 1-3 gray
__global__ void compute_tv_diffusivity_shared
(
 const float *d_input,
 float *d_output,
 int   nx,
 int   ny,
 size_t   pitch
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;
  const int idx = y*pitch + x;

  __shared__ float u[DIFF_BW+2][DIFF_BH+2];

  // load data into shared memory
  if (x < nx && y < ny) {

    u[tx][ty] = d_input[idx];

    if (x == 0)  u[0][ty] = u[tx][ty];
    else if (threadIdx.x == 0) u[0][ty] = d_input[idx-1];      
    if (x == nx-1) u[tx+1][ty] = u[tx][ty];
    else if (threadIdx.x == blockDim.x-1) u[tx+1][ty] = d_input[idx+1];

    if (y == 0)  u[tx][0] = u[tx][ty];
    else if (threadIdx.y == 0) u[tx][0] = d_input[idx-pitch];
    if (y == ny-1) u[tx][ty+1] = u[tx][ty];
    else if (threadIdx.y == blockDim.y-1) u[tx][ty+1] = d_input[idx+pitch];
  }

  __syncthreads();

 
  // make use of the constant TV_EPSILON
  float tempDerX;
  float tempDerY;
  float tmpGrad;
  if (x < nx && y < ny) {
	 tempDerX = 0.5f*(u[threadIdx.x + 2][threadIdx.y+1]-u[threadIdx.x][threadIdx.y+1]);
	 tempDerY = 0.5f*(u[threadIdx.x+1][threadIdx.y + 2] - u[threadIdx.x+1][threadIdx.y]);
	 tmpGrad = sqrt( tempDerX*tempDerX + tempDerY*tempDerY );
	 d_output[idx] = 1.0 / sqrt(tmpGrad*tmpGrad + TV_EPSILON);
  }
  // ### implement me ###
}


/*! Computes a joined diffusivity for an RGB Image:
 *  (g_R,g_G,g_B)(R,G,B) := 
 *  (g((R+G+B)/3),g((R+G+B)/3),g((R+G+B)/3))
 * */
__global__ void compute_tv_diffusivity_joined_shared
(
 const float3 *d_input,
 float3 *d_output,
 int   nx,
 int   ny,
 size_t   pitchBytes
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;
  const char* imgP = (char*)d_input + y*pitchBytes + x*sizeof(float3);

  __shared__ float3 u[DIFF_BW+2][DIFF_BH+2];

  // load data into shared memory
  if (x < nx && y < ny) {

    u[tx][ty] = *( (float3*)imgP );

    if (x == 0)  u[0][ty] = u[tx][ty];
    else if (threadIdx.x == 0) u[0][ty] = *( ((float3*)imgP)-1 );
    if (x == nx-1) u[tx+1][ty] = u[tx][ty];
    else if (threadIdx.x == blockDim.x-1) u[tx+1][ty] = *( ((float3*)imgP)+1 );

    if (y == 0)  u[tx][0] = u[tx][ty];
    else if (threadIdx.y == 0) u[tx][0] = *( (float3*)(imgP-pitchBytes) );
    if (y == ny-1) u[tx][ty+1] = u[tx][ty];
    else if (threadIdx.y == blockDim.y-1) u[tx][ty+1] = *( (float3*)(imgP+pitchBytes) );
  }

  __syncthreads();
  
  
  // make use of the constant TV_EPSILON
	float3 tmpGrad;
	float3 xValue;
	float3 yValue;
	if (x < nx && y < ny) {
		float avgr = ( u[threadIdx.x + 2][threadIdx.y+1].x + u[threadIdx.x + 2][threadIdx.y+1].y + u[threadIdx.x + 2][threadIdx.y+1].z ) / 3.0;
		float avgu = ( u[threadIdx.x+1][threadIdx.y + 2].x + u[threadIdx.x+1][threadIdx.y + 2].y + u[threadIdx.x+1][threadIdx.y + 2].z ) / 3.0;
		float avgl = ( u[threadIdx.x][threadIdx.y+1].x + u[threadIdx.x][threadIdx.y+1].y + u[threadIdx.x][threadIdx.y+1].z) / 3.0;
		float avgd = ( u[threadIdx.x+1][threadIdx.y].x + u[threadIdx.x+1][threadIdx.y].y + u[threadIdx.x+1][threadIdx.y].z) / 3.0;
		// x derivatives 
		xValue.x = 0.5f * (avgr	- avgl);
		xValue.y = xValue.x;
		xValue.z = xValue.x;
		
		// y derivatives 
		yValue.x = 0.5f * (avgu - avgd);
		yValue.y = yValue.x;
		yValue.z = yValue.x;
		
		float normX = sqrt(xValue.x*xValue.x + yValue.x*yValue.x);
		float normY = normX;
		float normZ = normX;
		
		tmpGrad.x = 1.0 / sqrt(normX*normX + TV_EPSILON);
		tmpGrad.y = 1.0 / sqrt(normY*normY + TV_EPSILON);
		tmpGrad.z = 1.0 / sqrt(normZ*normZ + TV_EPSILON);

		*((float3*)(((char*)d_output) + y*pitchBytes) + x) = tmpGrad;
	
	}
}


/*! Computes a separate diffusivity for an RGB Image:
 *  (g_R,g_G,g_B)(R,G,B) := 
 *  (g(R),g(G),g(B))
 * */
__global__ void compute_tv_diffusivity_separate_shared
(
 const float3 *d_input,
 float3 *d_output,
 int   nx,
 int   ny,
 size_t   pitchBytes
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;
  const char* imgP = (char*)d_input + y*pitchBytes + x*sizeof(float3);

  __shared__ float3 u[DIFF_BW+2][DIFF_BH+2];

  // load data into shared memory
  if (x < nx && y < ny) {

    u[tx][ty] = *( (float3*)imgP );

    if (x == 0)  u[threadIdx.x][ty] = u[tx][ty];
    else if (threadIdx.x == 0) u[0][ty] = *( ((float3*)imgP)-1 );
    if (x == nx-1) u[tx+1][ty] = u[tx][ty];
    else if (threadIdx.x == blockDim.x-1) u[tx+1][ty] = *( ((float3*)imgP)+1 );

    if (y == 0)  u[tx][0] = u[tx][ty];
    else if (threadIdx.y == 0) u[tx][0] = *( (float3*)(imgP-pitchBytes) );
    if (y == ny-1) u[tx][ty+1] = u[tx][ty];
    else if (threadIdx.y == blockDim.y-1) u[tx][ty+1] = *( (float3*)(imgP+pitchBytes) );
  }

  __syncthreads();

  
  // make use of the constant TV_EPSILON
	float3 tmpGrad;
	float3 xValue;
	float3 yValue;
	if (x < nx && y < ny) {
		// x derivatives 
		xValue.x = 0.5f * (u[threadIdx.x + 2][threadIdx.y+1].x
				- u[threadIdx.x][threadIdx.y+1].x);
		xValue.y = 0.5f * (u[threadIdx.x + 2][threadIdx.y+1].y
				- u[threadIdx.x][threadIdx.y+1].y);
		xValue.z = 0.5f * (u[threadIdx.x + 2][threadIdx.y+1].z
				- u[threadIdx.x][threadIdx.y+1].z);
		// y derivatives 
		yValue.x = 0.5f * (u[threadIdx.x+1][threadIdx.y + 2].x
				- u[threadIdx.x+1][threadIdx.y].x);
		yValue.y = 0.5f * (u[threadIdx.x+1][threadIdx.y + 2].y
				- u[threadIdx.x+1][threadIdx.y].y);
		yValue.z = 0.5f * (u[threadIdx.x+1][threadIdx.y + 2].z
				- u[threadIdx.x+1][threadIdx.y].z);
		
		float normX = sqrt(xValue.x*xValue.x + yValue.x*yValue.x);
		float normY = sqrt(xValue.y*xValue.y + yValue.y*yValue.y);
		float normZ = sqrt(xValue.z*xValue.z + yValue.z*yValue.z);
		
		tmpGrad.x = 1.0 / sqrt(normX*normX + TV_EPSILON);
		tmpGrad.y = 1.0 / sqrt(normY*normY + TV_EPSILON);
		tmpGrad.z = 1.0 / sqrt(normZ*normZ + TV_EPSILON);

		*((float3*)(((char*)d_output) + y*pitchBytes) + x) = tmpGrad;	
	}

}




//----------------------------------------------------------------------------
// Non-linear Diffusion - Jacobi scheme
//----------------------------------------------------------------------------



// mode 2 gray: Jacobi solver
__global__ void jacobi_shared
(
 float *d_output,
 const float *d_input,
 const float *d_original,
 const float *d_diffusivity,
 float weight,
 int   nx,
 int   ny,
 size_t   pitch
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int idx = y*pitch + x;

  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;

  __shared__ float u[DIFF_BW+2][DIFF_BH+2];
  __shared__ float g[DIFF_BW+2][DIFF_BH+2];


  // load data into shared memory
  if (x < nx && y < ny) {
    u[tx][ty] = d_input[idx];
    g[tx][ty] = d_diffusivity[idx];

    if (x == 0)  {
      u[0][ty] = u[tx][ty];
      g[0][ty] = g[tx][ty];
    }
    else if (threadIdx.x == 0) {
      u[0][ty] = d_input[idx-1];
      g[0][ty] = d_diffusivity[idx-1];
    }
    if (x == nx-1) {
      u[tx+1][ty] = u[tx][ty];
      g[tx+1][ty] = g[tx][ty];
    }
    else if (threadIdx.x == blockDim.x-1) {
      u[tx+1][ty] = d_input[idx+1];
      g[tx+1][ty] = d_diffusivity[idx+1];
    }

    if (y == 0) {
      u[tx][0] = u[tx][ty];
      g[tx][0] = g[tx][ty];
    }
    else if (threadIdx.y == 0) {
      u[tx][0] = d_input[idx-pitch];
      g[tx][0] = d_diffusivity[idx-pitch];
    }
    if (y == ny-1) {
      u[tx][ty+1] = u[tx][ty];
      g[tx][ty+1] = g[tx][ty];
    }
    else if (threadIdx.y == blockDim.y-1) {
      u[tx][ty+1] = d_input[idx+pitch];
      g[tx][ty+1] = d_diffusivity[idx+pitch];
    }
  }

  __syncthreads();

  
  // ### implement me ###

}



// mode 2 interleaved: Jacobi solver
__global__ void jacobi_shared
(
 float3 *d_output,
 const float3 *d_input,
 const float3 *d_original,
 const float3 *d_diffusivity,
 float weight,
 int   nx,
 int   ny,
 size_t   pitchBytes
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const char* imgP = (char*)d_input + y*pitchBytes + x*sizeof(float3);
  const char* diffP = (char*)d_diffusivity + y*pitchBytes + x*sizeof(float3);

  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;

  __shared__ float3 u[DIFF_BW+2][DIFF_BH+2];
  __shared__ float3 g[DIFF_BW+2][DIFF_BH+2];


  // load data into shared memory
  if (x < nx && y < ny) {
    u[tx][ty] = *( (float3*)imgP );
    g[tx][ty] = *( (float3*)diffP );

    if (x == 0)  {
      u[0][ty] = u[tx][ty];
      g[0][ty] = g[tx][ty];
    }
    else if (threadIdx.x == 0) {
      u[0][ty] = *( ((float3*)imgP)-1 );
      g[0][ty] = *( ((float3*)diffP)-1 );
    }
    if (x == nx-1) {
      u[tx+1][ty] = u[tx][ty];
      g[tx+1][ty] = g[tx][ty];
    }
    else if (threadIdx.x == blockDim.x-1) {
      u[tx+1][ty] = *( ((float3*)imgP)+1 );
      g[tx+1][ty] = *( ((float3*)diffP)+1 );
    }

    if (y == 0) {
      u[tx][0] = u[tx][ty];
      g[tx][0] = g[tx][ty];
    }
    else if (threadIdx.y == 0) {
      u[tx][0] = *( (float3*)(imgP-pitchBytes) );
      g[tx][0] = *( (float3*)(diffP-pitchBytes) );
    }
    if (y == ny-1) {
      u[tx][ty+1] = u[tx][ty];
      g[tx][ty+1] = g[tx][ty];
    }
    else if (threadIdx.y == blockDim.y-1) {
      u[tx][ty+1] = *( (float3*)(imgP+pitchBytes) );
      g[tx][ty+1] = *( (float3*)(diffP+pitchBytes) );
    }
  }

  __syncthreads();

  
  // ### implement me ###


}



//----------------------------------------------------------------------------
// Non-linear Diffusion - Successive Over-Relaxation (SOR)
//----------------------------------------------------------------------------



// mode 3 gray: SOR solver
__global__ void sor_shared
(
 float *d_output,
 const float *d_input,
 const float *d_original,
 const float *d_diffusivity,
 float weight,
 float overrelaxation,
 int   nx,
 int   ny,
 size_t   pitch,
 int   red
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int idx = y*pitch + x;
  
  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;

  __shared__ float u[DIFF_BW+2][DIFF_BH+2];
  __shared__ float g[DIFF_BW+2][DIFF_BH+2];


  // load data into shared memory
  if (x < nx && y < ny) {
    u[tx][ty] = d_input[idx];
    g[tx][ty] = d_diffusivity[idx];

    if (x == 0)  {
      u[0][ty] = u[tx][ty];
      g[0][ty] = g[tx][ty];
    }
    else if (threadIdx.x == 0) {
      u[0][ty] = d_input[idx-1];
      g[0][ty] = d_diffusivity[idx-1];
    }
    if (x == nx-1) {
      u[tx+1][ty] = u[tx][ty];
      g[tx+1][ty] = g[tx][ty];
    }
    else if (threadIdx.x == blockDim.x-1) {
      u[tx+1][ty] = d_input[idx+1];
      g[tx+1][ty] = d_diffusivity[idx+1];
    }

    if (y == 0) {
      u[tx][0] = u[tx][ty];
      g[tx][0] = g[tx][ty];
    }
    else if (threadIdx.y == 0) {
      u[tx][0] = d_input[idx-pitch];
      g[tx][0] = d_diffusivity[idx-pitch];
    }
    if (y == ny-1) {
      u[tx][ty+1] = u[tx][ty];
      g[tx][ty+1] = g[tx][ty];
    }
    else if (threadIdx.y == blockDim.y-1) {
      u[tx][ty+1] = d_input[idx+pitch];
      g[tx][ty+1] = d_diffusivity[idx+pitch];
    }
  }

  __syncthreads();


  // ### implement me ###

}



// mode 3 interleaved: SOR solver
__global__ void sor_shared
(
 float3 *d_output,
 const float3 *d_input,
 const float3 *d_original,
 const float3 *d_diffusivity,
 float weight,
 float overrelaxation,
 int   nx,
 int   ny,
 size_t   pitchBytes,
 int   red
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const char* imgP = (char*)d_input + y*pitchBytes + x*sizeof(float3);
  const char* diffP = (char*)d_diffusivity + y*pitchBytes + x*sizeof(float3);

  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;

  __shared__ float3 u[DIFF_BW+2][DIFF_BH+2];
  __shared__ float3 g[DIFF_BW+2][DIFF_BH+2];



  // load data into shared memory
  if (x < nx && y < ny) {
    u[tx][ty] = *( (float3*)imgP );
    g[tx][ty] = *( (float3*)diffP );

    if (x == 0)  {
      u[0][ty] = u[tx][ty];
      g[0][ty] = g[tx][ty];
    }
    else if (threadIdx.x == 0) {
      u[0][ty] = *( ((float3*)imgP)-1 );
      g[0][ty] = *( ((float3*)diffP)-1 );
    }
    if (x == nx-1) {
      u[tx+1][ty] = u[tx][ty];
      g[tx+1][ty] = g[tx][ty];
    }
    else if (threadIdx.x == blockDim.x-1) {
      u[tx+1][ty] = *( ((float3*)imgP)+1 );
      g[tx+1][ty] = *( ((float3*)diffP)+1 );
    }

    if (y == 0) {
      u[tx][0] = u[tx][ty];
      g[tx][0] = g[tx][ty];
    }
    else if (threadIdx.y == 0) {
      u[tx][0] = *( (float3*)(imgP-pitchBytes) );
      g[tx][0] = *( (float3*)(diffP-pitchBytes) );
    }
    if (y == ny-1) {
      u[tx][ty+1] = u[tx][ty];
      g[tx][ty+1] = g[tx][ty];
    }
    else if (threadIdx.y == blockDim.y-1) {
      u[tx][ty+1] = *( (float3*)(imgP+pitchBytes) );
      g[tx][ty+1] = *( (float3*)(diffP+pitchBytes) );
    }
  }

  __syncthreads();

  
  // ### implement me ###


}




//----------------------------------------------------------------------------
// Host function
//----------------------------------------------------------------------------



void gpu_diffusion
(
 const float *input,
 float *output,
 int nx, int ny, int nc, 
 float timeStep,
 int iterations,
 float weight,
 int lagged_iterations,
 float overrelaxation,
 int mode,
 bool jointDiffusivity
 )
{
  int i,j;
  size_t pitchF1, pitchBytesF1, pitchBytesF3;
  float *d_input = 0;
  float *d_output = 0;
  float *d_diffusivity = 0;
  float *d_original = 0;
  float *temp = 0;

  dim3 dimGrid((int)ceil((float)nx/DIFF_BW), (int)ceil((float)ny/DIFF_BH));
  dim3 dimBlock(DIFF_BW,DIFF_BH);

  // Allocation of GPU Memory
  if (nc == 1) {

    cutilSafeCall( hipMallocPitch( (void**)&(d_input), &pitchBytesF1, nx*sizeof(float), ny ) );
    cutilSafeCall( hipMallocPitch( (void**)&(d_output), &pitchBytesF1, nx*sizeof(float), ny ) );
    if (mode) cutilSafeCall( hipMallocPitch( (void**)&(d_diffusivity), &pitchBytesF1, nx*sizeof(float), ny ) );
    if (mode >= 2) cutilSafeCall( hipMallocPitch( (void**)&(d_original), &pitchBytesF1, nx*sizeof(float), ny ) );

    cutilSafeCall( hipMemcpy2D(d_input, pitchBytesF1, input, nx*sizeof(float), nx*sizeof(float), ny, hipMemcpyHostToDevice) );
    if (mode >= 2) cutilSafeCall( hipMemcpy2D(d_original, pitchBytesF1, d_input, pitchBytesF1, nx*sizeof(float), ny, hipMemcpyDeviceToDevice) );

    pitchF1 = pitchBytesF1/sizeof(float);

  } else if (nc == 3) {

    cutilSafeCall( hipMallocPitch( (void**)&(d_input), &pitchBytesF3, nx*sizeof(float3), ny ) );
    cutilSafeCall( hipMallocPitch( (void**)&(d_output), &pitchBytesF3, nx*sizeof(float3), ny ) );
    if (mode) cutilSafeCall( hipMallocPitch( (void**)&(d_diffusivity), &pitchBytesF3, nx*sizeof(float3), ny ) );
    if (mode >= 2) cutilSafeCall( hipMallocPitch( (void**)&(d_original), &pitchBytesF3, nx*sizeof(float3), ny ) );

    cutilSafeCall( hipMemcpy2D(d_input, pitchBytesF3, input, nx*sizeof(float3), nx*sizeof(float3), ny, hipMemcpyHostToDevice) );
    if (mode >= 2) cutilSafeCall( hipMemcpy2D(d_original, pitchBytesF3, d_input, pitchBytesF3, nx*sizeof(float3), ny, hipMemcpyDeviceToDevice) );

  }


  // Execution of the Diffusion Kernel

  if (mode == 0) {   // linear isotropic diffision
    if (nc == 1) {
      for (i=0;i<iterations;i++) {
        diffuse_linear_isotrop_shared<<<dimGrid,dimBlock>>>(d_input, d_output, timeStep, nx, ny, pitchF1);

        cutilSafeCall( hipDeviceSynchronize() );

        temp = d_input;
        d_input = d_output;
        d_output = temp;
      }
    }
    else if (nc == 3) {
      for (i=0;i<iterations;i++) {
        diffuse_linear_isotrop_shared<<<dimGrid,dimBlock>>>((float3*)d_input,(float3*)d_output,timeStep,nx,ny,pitchBytesF3);

        cutilSafeCall( hipDeviceSynchronize() );

        temp = d_input;
        d_input = d_output;
        d_output = temp;
      }
    }
  }
  else if (mode == 1) {  // nonlinear isotropic diffusion
    if (nc == 1) {

      for (i=0;i<iterations;i++) {
        compute_tv_diffusivity_shared<<<dimGrid,dimBlock>>>(d_input,d_diffusivity,nx,ny,pitchF1);

        cutilSafeCall( hipDeviceSynchronize() );

        diffuse_nonlinear_isotrop_shared<<<dimGrid,dimBlock>>>(d_input,d_diffusivity,d_output,timeStep,nx,ny,pitchF1);

        cutilSafeCall( hipDeviceSynchronize() );

        temp = d_input;
        d_input = d_output;
        d_output = temp;
      }
    }
    else if (nc == 3) {
      for (i=0;i<iterations;i++) {
        if (jointDiffusivity)
          compute_tv_diffusivity_joined_shared<<<dimGrid,dimBlock>>>((float3*)d_input,(float3*)d_diffusivity,nx,ny,pitchBytesF3);
        else
          compute_tv_diffusivity_separate_shared<<<dimGrid,dimBlock>>>((float3*)d_input,(float3*)d_diffusivity,nx,ny,pitchBytesF3);


        cutilSafeCall( hipDeviceSynchronize() );

        diffuse_nonlinear_isotrop_shared<<<dimGrid,dimBlock>>>
          ((float3*)d_input,(float3*)d_diffusivity,(float3*)d_output,timeStep,nx,ny,pitchBytesF3);

        cutilSafeCall( hipDeviceSynchronize() );

        temp = d_input;
        d_input = d_output;
        d_output = temp;
      }
    }
  }
  else if (mode == 2) {    // Jacobi-method
    if (nc == 1) {
      for (i=0;i<iterations;i++) {
        compute_tv_diffusivity_shared<<<dimGrid,dimBlock>>>(d_input,d_diffusivity,nx,ny,pitchF1);

        cutilSafeCall( hipDeviceSynchronize() );

        for (j=0;j<lagged_iterations;j++) {
          jacobi_shared<<<dimGrid,dimBlock>>> (d_output,d_input,d_original,
            d_diffusivity,weight,nx,ny,pitchF1);

          cutilSafeCall( hipDeviceSynchronize() );

          temp = d_input;
          d_input = d_output;
          d_output = temp;
        }
      }
    }
    else if (nc == 3) {
      for (i=0;i<iterations;i++) {
        if (jointDiffusivity)
          compute_tv_diffusivity_joined_shared<<<dimGrid,dimBlock>>>((float3*)d_input,(float3*)d_diffusivity,nx,ny,pitchBytesF3);
        else
          compute_tv_diffusivity_separate_shared<<<dimGrid,dimBlock>>>((float3*)d_input,(float3*)d_diffusivity,nx,ny,pitchBytesF3);

        cutilSafeCall( hipDeviceSynchronize() );

        for (j=0;j<lagged_iterations;j++) {
          jacobi_shared<<<dimGrid,dimBlock>>>
            ((float3*)d_output,(float3*)d_input,
            (float3*)d_original,(float3*)d_diffusivity,
            weight,nx,ny,pitchBytesF3);

          cutilSafeCall( hipDeviceSynchronize() );

          temp = d_input;
          d_input = d_output;
          d_output = temp;
        }
      }
    }    
  }
  else if (mode == 3) {    // Successive Over Relaxation (Gauss-Seidel with extrapolation)
    if (nc == 1) {
      for (i=0;i<iterations;i++) {
        compute_tv_diffusivity_shared<<<dimGrid,dimBlock>>>(d_input,d_diffusivity,nx,ny,pitchF1);

        cutilSafeCall( hipDeviceSynchronize() );

        for(j=0;j<lagged_iterations;j++) {					
          sor_shared<<<dimGrid,dimBlock>>>(d_input,d_input,d_original,
            d_diffusivity,weight,overrelaxation,nx,ny,pitchF1, 0);

          cutilSafeCall( hipDeviceSynchronize() );

          sor_shared<<<dimGrid,dimBlock>>>(d_input,d_input,d_original,
            d_diffusivity,weight,overrelaxation,nx,ny,pitchF1, 1);

          cutilSafeCall( hipDeviceSynchronize() );
        }
      }
    }
    if (nc == 3) {
      for (i=0;i<iterations;i++) {
        if (jointDiffusivity)
          compute_tv_diffusivity_joined_shared<<<dimGrid,dimBlock>>>((float3*)d_input,(float3*)d_diffusivity,nx,ny,pitchBytesF3);
        else
          compute_tv_diffusivity_separate_shared<<<dimGrid,dimBlock>>>((float3*)d_input,(float3*)d_diffusivity,nx,ny,pitchBytesF3);

        cutilSafeCall( hipDeviceSynchronize() );

        for (j=0;j<lagged_iterations;j++) {
          sor_shared<<<dimGrid,dimBlock>>>
            ((float3*)d_input,(float3*)d_input,
            (float3*)d_original,(float3*)d_diffusivity,
            weight,overrelaxation,nx,ny,pitchBytesF3, 0);

          cutilSafeCall( hipDeviceSynchronize() );

          sor_shared<<<dimGrid,dimBlock>>>
            ((float3*)d_input,(float3*)d_input,
            (float3*)d_original,(float3*)d_diffusivity,
            weight,overrelaxation,nx,ny,pitchBytesF3, 1);

          cutilSafeCall( hipDeviceSynchronize() );
        }
      }
    }
  }


  if (nc == 1) {
    if (mode == 3) cutilSafeCall( hipMemcpy2D(output, nx*sizeof(float), d_input, pitchBytesF1, nx*sizeof(float), ny, hipMemcpyDeviceToHost) );
    else cutilSafeCall( hipMemcpy2D(output, nx*sizeof(float), d_output, pitchBytesF1, nx*sizeof(float), ny, hipMemcpyDeviceToHost) );
  } else if (nc == 3) {
    if (mode == 3) cutilSafeCall( hipMemcpy2D(output, nx*sizeof(float3), d_input, pitchBytesF3, nx*sizeof(float3), ny, hipMemcpyDeviceToHost) );
    else cutilSafeCall( hipMemcpy2D(output, nx*sizeof(float3), d_output, pitchBytesF3, nx*sizeof(float3), ny, hipMemcpyDeviceToHost) );
  }


  // clean up
  if (d_original) cutilSafeCall( hipFree(d_original) );
  if (d_diffusivity) cutilSafeCall( hipFree(d_diffusivity) );
  if (d_output) cutilSafeCall( hipFree(d_output) );
  if (d_input)  cutilSafeCall( hipFree(d_input) );
}